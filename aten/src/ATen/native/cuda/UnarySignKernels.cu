#include "hip/hip_runtime.h"
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Math.cuh>
#include <ATen/native/cuda/zmath.cuh>

namespace at { namespace native {

void logical_not_kernel_cuda(TensorIterator& iter) {
  // error check -- this is just ensuring we don't dispatch on types that aren't in ALL_TYPES_AND2(...)
  // so we don't have to maintain a separate list or to do double dispatch.
  AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, iter.dtype(0), "logical_not_cuda", [&]() {});

  AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, iter.dtype(1), "logical_not_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> bool { return !a; });
  });
}

void neg_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "neg_cuda", [&]() {
    AT_SKIP_BFLOAT16_IF_NOT_ROCM(scalar_t, "neg_cuda", [&] {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return -a;
      });
    });
  });
}

void sign_kernel_cuda(TensorIterator& iter){
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel(iter, []GPU_LAMBDA(bool a){
      return a;
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND(ScalarType::Half, iter.dtype(), "sign_cuda", [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
            scalar_t zero = scalar_t(0);
            return (zero < a) - (a < zero);
        });
    });
  }
}

template<typename T>
__host__ __device__ static inline thrust::complex<T> sgn_wrapper(thrust::complex<T> v) {
  if (v == thrust::complex<T>(0, 0)) {
    return thrust::complex<T>(0, 0);
  } else {
    return z / std::abs(z);
  }
}

void sgn_kernel_cuda(TensorIterator& iter){
  AT_DISPATCH_COMPLEX_TYPES(iter.dtype(), "sgn_cuda", [&]() {
      using thrust_t = typename ztype_cuda<scalar_t>::thrust_t;
      gpu_kernel(iter, []GPU_LAMBDA(thrust_t a) -> thrust_t {
        return sgn_wrapper(a);
      });
  });
}

REGISTER_DISPATCH(logical_not_stub, &logical_not_kernel_cuda);
REGISTER_DISPATCH(neg_stub, &neg_kernel_cuda);
REGISTER_DISPATCH(sign_stub, &sign_kernel_cuda);
REGISTER_DISPATCH(sgn_stub, &sgn_kernel_cuda);

}} // namespace at::native
